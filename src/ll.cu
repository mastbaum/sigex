#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "ll.h"

// macro to print error and abort on cuda errors, c/o stan
#define CUDA_CHECK_ERROR(call) do { \
  hipError_t err = call; \
  if (hipSuccess != err) { \
    fprintf(stderr, "Cuda error in file '%s' in line %i: %s.\n", \
            __FILE__, __LINE__, hipGetErrorString(err)); \
    exit(EXIT_FAILURE); \
  } \
} while (0)


// compute the log likelihood given signal rates and normalizations
__global__ void ll(const float* lut, const float* n, const double* pars, const size_t ne, const size_t ns, double* sums) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  for (int i=idx; i<(int)ne; i+=gridDim.x*blockDim.x) {
    double s = 0;
    for (size_t j=0; j<ns; j++) {
      s += pars[j] * n[j] * lut[i*ns+j];
    }
    sums[idx] += log(s);
  }
}


GPULL::GPULL(const float* lut, const size_t _nsignals, const size_t _nevents)
  : nsignals(_nsignals), nevents(_nevents), norms_device_alloc(false) {
  CUDA_CHECK_ERROR(hipMalloc(&this->lut_device, this->nevents * this->nsignals * sizeof(float)));
  CUDA_CHECK_ERROR(hipMemcpy(this->lut_device, lut, this->nevents * this->nsignals * sizeof(float), hipMemcpyHostToDevice));
}


GPULL::~GPULL() {
  CUDA_CHECK_ERROR(hipFree(this->norms_device));
  CUDA_CHECK_ERROR(hipFree(this->lut_device));
}


void GPULL::set_norms(const float* norms, const size_t _nsignals) {
  if (!this->norms_device_alloc) {
    CUDA_CHECK_ERROR(hipMalloc(&this->norms_device, this->nsignals * sizeof(float)));
    this->norms_device_alloc = true;
  }
  CUDA_CHECK_ERROR(hipMemcpy(this->norms_device, norms, this->nsignals * sizeof(float), hipMemcpyHostToDevice));
}


double GPULL::operator()(const double* pars) {
  size_t blocksize = 256;
  size_t nblocks = 16;
  size_t nthreads = nblocks * blocksize;

  double* pars_device;
  CUDA_CHECK_ERROR(hipMalloc(&pars_device, this->nsignals * sizeof(double)));
  CUDA_CHECK_ERROR(hipMemcpy(pars_device, pars, this->nsignals * sizeof(double), hipMemcpyHostToDevice));

  double* sums_device;
  CUDA_CHECK_ERROR(hipMalloc(&sums_device, nthreads * sizeof(double)));
  CUDA_CHECK_ERROR(hipMemset(sums_device, 0, nthreads * sizeof(double)));

  ll<<<nblocks, blocksize>>>(this->lut_device, this->norms_device, pars_device, this->nevents, this->nsignals, sums_device);
  CUDA_CHECK_ERROR(hipDeviceSynchronize());

  double* sums = new double[blocksize * nblocks];
  CUDA_CHECK_ERROR(hipMemcpy(sums, sums_device, nthreads * sizeof(double), hipMemcpyDeviceToHost));

  CUDA_CHECK_ERROR(hipFree(pars_device));
  CUDA_CHECK_ERROR(hipFree(sums_device));

  double sum = 0;
  for (size_t i=0; i<nthreads; i++) {
    sum += sums[i];
  }

  delete[] sums;

  return sum;
}

